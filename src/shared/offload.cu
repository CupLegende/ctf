
#include <hip/hip_runtime.h>
/*Copyright (c) 2014, Edgar Solomonik, all rights reserved.*/

#ifdef USE_CUDA
#include <cuda_runtime.h>
#include <cublas_v2.h>
#include <complex>
#include <assert.h>
#include <stdio.h>
//#include "../shared/util.h"
#include "device_launch_parameters.h"

template<typename dtype>
dtype get_zero(){
  assert(0);
}
template<> inline
double get_zero<double>() { return 0.0; }

template<> inline
std::complex<double> get_zero< std::complex<double> >() { return std::complex<double>(0.0,0.0); }


typedef int64_t long_int;
#ifndef LIBT_ASSERT
#ifdef DEBUG
#define LIBT_ASSERT(...)                \
do { assert(__VA_ARGS__); } while (0)
#else
#define LIBT_ASSERT(...) do {} while(0 && (__VA_ARGS__))
#endif
#endif

#include "offload.h"

int initialized = 0;
cublasHandle_t cuhandle;

void offload_init(){
  if (!initialized){
    int ndev=0;
    cudaGetDeviceCount(&ndev);
    LIBT_ASSERT(ndev > 0);
    cublasStatus_t status = cublasCreate(&cuhandle);
    LIBT_ASSERT(status == CUBLAS_STATUS_SUCCESS);
  }
  initialized = 1;
}

void offload_exit(){
  if (initialized){
    cublasStatus_t status = cublasDestroy(cuhandle);
    LIBT_ASSERT(status == CUBLAS_STATUS_SUCCESS);
    initialized = 0;
  }
}


/**
 * \brief allocates offload device pointer
 * \param[in] size number of elements to create for buffer
 */
template <typename dtype>
offload_ptr<dtype>::offload_ptr(long_int size_){
  size = size_;
  cudaError_t err = cudaMalloc((void**)&dev_ptr, size_*sizeof(dtype));
  LIBT_ASSERT(err == cudaSuccess);
}

/**
 * \brief deallocates offload device pointer
 */
template <typename dtype>
offload_ptr<dtype>::~offload_ptr(){
  cudaError_t err = cudaFree(dev_ptr);
  LIBT_ASSERT(err == cudaSuccess);
}

/**
 * \brief downloads all data from device pointer to host pointer
 * \param[in,out] host_ptr preallocated host buffer to download to
 */
template <typename dtype>
void offload_ptr<dtype>::download(dtype * host_ptr){
  cudaError_t err = cudaMemcpy(host_ptr, dev_ptr, size*sizeof(dtype),
                               cudaMemcpyDeviceToHost);
  LIBT_ASSERT(err == cudaSuccess);
}
/**
 * \brief uploads all data to device pointer from host pointer
 * \param[in] host_ptr preallocated host buffer to upload from
 */
template <typename dtype>
void offload_ptr<dtype>::upload(dtype const * host_ptr){
  cudaError_t err = cudaMemcpy(dev_ptr, host_ptr, size*sizeof(dtype),
                               cudaMemcpyHostToDevice);
  LIBT_ASSERT(err == cudaSuccess);
}


template <typename dtype>
__global__ void gset_zero(dtype *arr, int64_t size, dtype val) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i=idx; i<size; i+= gridDim.x*blockDim.x) {
    arr[i]=val;
  }
}

/**
 * \brief set array to 0
 */
template <typename dtype>
void offload_ptr<dtype>::set_zero(){
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / (size);
  gset_zero<<<blockSize, numBlocks>>>(dev_ptr, size, get_zero<dtype>());
}


void host_pinned_alloc(void ** ptr, long_int size){
  cudaError_t err = cudaHostAlloc(ptr, size, cudaHostAllocMapped);
  LIBT_ASSERT(err == cudaSuccess);
}

void host_pinned_free(void * ptr){
  cudaError_t err = cudaFreeHost(ptr);
  LIBT_ASSERT(err == cudaSuccess);
}

/**
 * \brief performs an offloaded gemm using device pointer of objects
 *        specialized instantization to double
 */
template <typename dtype>
void offload_gemm(char                  tA,
                  char                  tB,
                  int                   m,
                  int                   n,
                  int                   k,
                  dtype                 alpha,
                  offload_ptr<dtype> &  A,
                  int                   lda_A,
                  offload_ptr<dtype> &  B,
                  int                   lda_B,
                  dtype                 beta,
                  offload_ptr<dtype> &  C,
                  int                   lda_C){
  offload_gemm(tA, tB, m, n, k, alpha, A.dev_ptr, lda_A, B.dev_ptr, lda_B, beta, C.dev_ptr, lda_C);
}
template 
void offload_gemm(char                  tA,
                  char                  tB,
                  int                   m,
                  int                   n,
                  int                   k,
                  double                alpha,
                  offload_ptr<double> & A,
                  int                   lda_A,
                  offload_ptr<double> & B,
                  int                   lda_B,
                  double                beta,
                  offload_ptr<double> & C,
                  int                   lda_C);
template 
void offload_gemm(char                                  tA,
                  char                                  tB,
                  int                                   m,
                  int                                   n,
                  int                                   k,
                  std::complex<double>                  alpha,
                  offload_ptr< std::complex<double> > & A,
                  int                                   lda_A,
                  offload_ptr< std::complex<double> > & B,
                  int                                   lda_B,
                  std::complex<double>                  beta,
                  offload_ptr< std::complex<double> > & C,
                  int                                   lda_C);

/**
 * \brief performs an offloaded gemm using device pointer of objects
 *        specialized instantization to double
 */
template <>
void offload_gemm<double>(char                  tA,
                          char                  tB,
                          int                   m,
                          int                   n,
                          int                   k,
                          double                alpha,
                          double const        * dev_A,
                          int                   lda_A,
                          double const        * dev_B,
                          int                   lda_B,
                          double                beta,
                          double              * dev_C,
                          int                   lda_C){
  LIBT_ASSERT(initialized);

  cublasOperation_t cuA;  
  switch (tA){
    case 'n':
    case 'N':
      cuA = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuA = CUBLAS_OP_T;
      break;
  }  

  cublasOperation_t cuB;
  switch (tB){
    case 'n':
    case 'N':
      cuB = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuB = CUBLAS_OP_T;
      break;
  }  

  //printf("offloading dgemm\n");
  cublasStatus_t status = 
    cublasDgemm(cuhandle, cuA, cuB, m, n, k, &alpha, 
                dev_A, lda_A, 
                dev_B, lda_B, &beta, 
                dev_C, lda_C);
  
  LIBT_ASSERT(status == CUBLAS_STATUS_SUCCESS);
}

/**
 * \brief performs an offloaded gemm using device pointer of objects
 *        specialized instantization to complex<double>
 */
template <>
void offload_gemm< std::complex<double> >(
                         char                                  tA,
                         char                                  tB,
                         int                                   m,
                         int                                   n,
                         int                                   k,
                         std::complex<double>                  alpha,
                         std::complex<double> const          * dev_A,
                         int                                   lda_A,
                         std::complex<double> const          * dev_B,
                         int                                   lda_B,
                         std::complex<double>                  beta,
                         std::complex<double>                * dev_C,
                         int                                   lda_C){
  LIBT_ASSERT(initialized);
  
  cublasOperation_t cuA;  
  switch (tA){
    case 'n':
    case 'N':
      cuA = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuA = CUBLAS_OP_T;
      break;
    case 'c':
    case 'C':
      cuA = CUBLAS_OP_C;
      break;
  }  

  cublasOperation_t cuB;
  switch (tB){
    case 'n':
    case 'N':
      cuB = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuB = CUBLAS_OP_T;
      break;
    case 'c':
    case 'C':
      cuB = CUBLAS_OP_C;
      break;
  }  

  //printf("offloading zgemm\n");
  cublasStatus_t status = 
    cublasZgemm(cuhandle, cuA, cuB, m, n, k, 
                reinterpret_cast<cuDoubleComplex*>(&alpha), 
                reinterpret_cast<const cuDoubleComplex*>(dev_A), lda_A, 
                reinterpret_cast<const cuDoubleComplex*>(dev_B), lda_B, 
                reinterpret_cast<cuDoubleComplex*>(&beta), 
                reinterpret_cast<cuDoubleComplex*>(dev_C), lda_C);
  
  LIBT_ASSERT(status == CUBLAS_STATUS_SUCCESS);
}

template class offload_ptr<double>;
template class offload_ptr< std::complex<double> >;
#endif


#include <hip/hip_runtime.h>
/*Copyright (c) 2014, Edgar Solomonik, all rights reserved.*/

#ifdef USE_CUDA
#include <cuda_runtime.h>
#include <cublas_v2.h>
#include <complex>
#include <assert.h>
#include <stdio.h>
//#include "../shared/util.h"
#include "device_launch_parameters.h"
#include "int_timer.h"

typedef int64_t int64_t;
volatile static int64_t int64_t_max = INT64_MAX;
#include "offload.h"

#ifndef ASSERT
#if ENABLE_ASSERT
#define ASSERT(...)                \
do { if (!(__VA_ARGS__)) handler(); assert(__VA_ARGS__); } while (0)
#else
#define ASSERT(...) do {} while(0 && (__VA_ARGS__))
#endif
#endif

#ifndef PROFILE
#define TAU_PROFILE(NAME,ARG,USER)
#define TAU_PROFILE_TIMER(ARG1, ARG2, ARG3, ARG4)
#define TAU_PROFILER_CREATE(ARG1, ARG2, ARG3, ARG4)
#define TAU_PROFILE_STOP(ARG)
#define TAU_PROFILE_START(ARG)
#define TAU_PROFILE_SET_NODE(ARG)
#define TAU_PROFILE_SET_CONTEXT(ARG)
#define TAU_FSTART(ARG)
#define TAU_FSTOP(ARG)
#endif

#define ABORT                                   \
  do{                                           \
   assert(0); } while (0)

int initialized = 0;
cublasHandle_t cuhandle;

void offload_init(){
  if (!initialized){
    int ndev=0;
    cudaGetDeviceCount(&ndev);
    ASSERT(ndev > 0);
    cublasStatus_t status = cublasCreate(&cuhandle);
    ASSERT(status == CUBLAS_STATUS_SUCCESS);
  }
  initialized = 1;
}

void offload_exit(){
  if (initialized){
    cublasStatus_t status = cublasDestroy(cuhandle);
    ASSERT(status == CUBLAS_STATUS_SUCCESS);
    initialized = 0;
  }
}

template <typename dtype>
offload_ptr<dtype>::offload_ptr(int el_size_, int64_t size_){
  el_size = el_size_;
  size = size_;
  cudaError_t err = cudaMalloc((void**)&dev_ptr, size_*el_size);
  ASSERT(err == cudaSuccess);
}

offload_ptr::~offload_ptr(){
  cudaError_t err = cudaFree(dev_ptr);
  ASSERT(err == cudaSuccess);
}

void offload_ptr::download(dtype * host_ptr){
  TAU_FSTART(cuda_download);
  cudaError_t err = cudaMemcpy(host_ptr, dev_ptr, size*el_size,
                               cudaMemcpyDeviceToHost);
  TAU_FSTOP(cuda_download);
  ASSERT(err == cudaSuccess);
}

void offload_ptr<dtype>::upload(char const * host_ptr){
  TAU_FSTART(cuda_upload);
  cudaError_t err = cudaMemcpy(dev_ptr, host_ptr, size*el_size,
                               cudaMemcpyHostToDevice);
  TAU_FSTOP(cuda_upload);
  ASSERT(err == cudaSuccess);
}

/*
template <typename dtype>
__global__ void gset_zero(dtype *arr, int64_t size, dtype val) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i=idx; i<size; i+= gridDim.x*blockDim.x) {
    arr[i]=val;
  }
}

void offload_ptr<dtype>::set_zero(){
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / (size);
  gset_zero<<<blockSize, numBlocks>>>(dev_ptr, size, get_zero<dtype>());
}
*/

void host_pinned_alloc(void ** ptr, int64_t size){
  cudaError_t err = cudaHostAlloc(ptr, size, cudaHostAllocMapped);
  ASSERT(err == cudaSuccess);
}

void host_pinned_free(void * ptr){
  cudaError_t err = cudaFreeHost(ptr);
  ASSERT(err == cudaSuccess);
}

/*template <typename dtype>
void offload_gemm(char                  tA,
                  char                  tB,
                  int                   m,
                  int                   n,
                  int                   k,
                  dtype                 alpha,
                  offload_ptr<dtype> &  A,
                  int                   lda_A,
                  offload_ptr<dtype> &  B,
                  int                   lda_B,
                  dtype                 beta,
                  offload_ptr<dtype> &  C,
                  int                   lda_C){
  TAU_FSTART(cuda_gemm);
  offload_gemm(tA, tB, m, n, k, alpha, A.dev_ptr, lda_A, B.dev_ptr, lda_B, beta, C.dev_ptr, lda_C);
  TAU_FSTOP(cuda_gemm);
}
template 
void offload_gemm(char                  tA,
                  char                  tB,
                  int                   m,
                  int                   n,
                  int                   k,
                  double                alpha,
                  offload_ptr<double> & A,
                  int                   lda_A,
                  offload_ptr<double> & B,
                  int                   lda_B,
                  double                beta,
                  offload_ptr<double> & C,
                  int                   lda_C);
template 
void offload_gemm(char                                  tA,
                  char                                  tB,
                  int                                   m,
                  int                                   n,
                  int                                   k,
                  std::complex<double>                  alpha,
                  offload_ptr< std::complex<double> > & A,
                  int                                   lda_A,
                  offload_ptr< std::complex<double> > & B,
                  int                                   lda_B,
                  std::complex<double>                  beta,
                  offload_ptr< std::complex<double> > & C,
                  int                                   lda_C);
template <>
void offload_gemm<double>(char                  tA,
                          char                  tB,
                          int                   m,
                          int                   n,
                          int                   k,
                          double                alpha,
                          double const        * dev_A,
                          int                   lda_A,
                          double const        * dev_B,
                          int                   lda_B,
                          double                beta,
                          double              * dev_C,
                          int                   lda_C){
  ASSERT(initialized);

  cublasOperation_t cuA;  
  switch (tA){
    case 'n':
    case 'N':
      cuA = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuA = CUBLAS_OP_T;
      break;
  }  

  cublasOperation_t cuB;
  switch (tB){
    case 'n':
    case 'N':
      cuB = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuB = CUBLAS_OP_T;
      break;
  }  

  //printf("offloading dgemm\n");
  cublasStatus_t status = 
    cublasDgemm(cuhandle, cuA, cuB, m, n, k, &alpha, 
                dev_A, lda_A, 
                dev_B, lda_B, &beta, 
                dev_C, lda_C);
#ifdef PROFILE
  cudaDeviceSynchronize();
#endif
  
  ASSERT(status == CUBLAS_STATUS_SUCCESS);
}
*/

/*
template <>
void offload_gemm< std::complex<double> >(
                         char                                  tA,
                         char                                  tB,
                         int                                   m,
                         int                                   n,
                         int                                   k,
                         std::complex<double>                  alpha,
                         std::complex<double> const          * dev_A,
                         int                                   lda_A,
                         std::complex<double> const          * dev_B,
                         int                                   lda_B,
                         std::complex<double>                  beta,
                         std::complex<double>                * dev_C,
                         int                                   lda_C){
  ASSERT(initialized);
  
  cublasOperation_t cuA;  
  switch (tA){
    case 'n':
    case 'N':
      cuA = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuA = CUBLAS_OP_T;
      break;
    case 'c':
    case 'C':
      cuA = CUBLAS_OP_C;
      break;
  }  

  cublasOperation_t cuB;
  switch (tB){
    case 'n':
    case 'N':
      cuB = CUBLAS_OP_N;
      break;
    case 't':
    case 'T':
      cuB = CUBLAS_OP_T;
      break;
    case 'c':
    case 'C':
      cuB = CUBLAS_OP_C;
      break;
  }  

  TAU_FSTART(cublas_zgemm);
  cublasStatus_t status = 
    cublasZgemm(cuhandle, cuA, cuB, m, n, k, 
                reinterpret_cast<cuDoubleComplex*>(&alpha), 
                reinterpret_cast<const cuDoubleComplex*>(dev_A), lda_A, 
                reinterpret_cast<const cuDoubleComplex*>(dev_B), lda_B, 
                reinterpret_cast<cuDoubleComplex*>(&beta), 
                reinterpret_cast<cuDoubleComplex*>(dev_C), lda_C);
#ifdef PROFILE
  cudaDeviceSynchronize();
#endif
  TAU_FSTOP(cublas_zgemm);
  
  ASSERT(status == CUBLAS_STATUS_SUCCESS);
  assert(status == CUBLAS_STATUS_SUCCESS);
}
*/
#endif
